/*
dive into memory copy between host(CPU memory) and device(GPU memory)
demonstrate that pinned memory is 2.5x faster than pagable memory

compile: make

run: ./test

output(using 2080ti):

#################### pagable memory #########################

Data Length is 104857600
========= Test Host To Device ========
Step  0 Time is 20.865 ms, Bandwidth is 5025.526 MB/s
Step  1 Time is 20.749 ms, Bandwidth is 5053.622 MB/s
Step  2 Time is 20.587 ms, Bandwidth is 5093.389 MB/s
Step  3 Time is 20.632 ms, Bandwidth is 5082.280 MB/s
Step  4 Time is 20.585 ms, Bandwidth is 5093.884 MB/s
Step  5 Time is 20.483 ms, Bandwidth is 5119.250 MB/s
Step  6 Time is 20.725 ms, Bandwidth is 5059.474 MB/s
Step  7 Time is 20.452 ms, Bandwidth is 5127.010 MB/s
Step  8 Time is 20.822 ms, Bandwidth is 5035.904 MB/s
Step  9 Time is 20.605 ms, Bandwidth is 5088.940 MB/s
Step 10 Time is 20.674 ms, Bandwidth is 5071.955 MB/s
Step 11 Time is 20.755 ms, Bandwidth is 5052.161 MB/s
Step 12 Time is 20.728 ms, Bandwidth is 5058.742 MB/s
Step 13 Time is 20.696 ms, Bandwidth is 5066.564 MB/s
Step 14 Time is 20.749 ms, Bandwidth is 5053.622 MB/s
Step 15 Time is 20.871 ms, Bandwidth is 5024.081 MB/s
Step 16 Time is 20.605 ms, Bandwidth is 5088.940 MB/s
Step 17 Time is 20.635 ms, Bandwidth is 5081.541 MB/s
Step 18 Time is 20.775 ms, Bandwidth is 5047.297 MB/s
Step 19 Time is 20.536 ms, Bandwidth is 5106.038 MB/s
========= Test Device To Host ========
Step  0 Time is 19.895 ms, Bandwidth is 5270.550 MB/s
Step  1 Time is 21.020 ms, Bandwidth is 4988.468 MB/s
Step  2 Time is 21.183 ms, Bandwidth is 4950.083 MB/s
Step  3 Time is 20.933 ms, Bandwidth is 5009.201 MB/s
Step  4 Time is 20.994 ms, Bandwidth is 4994.646 MB/s
Step  5 Time is 21.194 ms, Bandwidth is 4947.513 MB/s
Step  6 Time is 20.965 ms, Bandwidth is 5001.555 MB/s
Step  7 Time is 21.064 ms, Bandwidth is 4978.048 MB/s
Step  8 Time is 21.220 ms, Bandwidth is 4941.451 MB/s
Step  9 Time is 21.179 ms, Bandwidth is 4951.018 MB/s
Step 10 Time is 21.107 ms, Bandwidth is 4967.906 MB/s
Step 11 Time is 20.881 ms, Bandwidth is 5021.675 MB/s
Step 12 Time is 21.053 ms, Bandwidth is 4980.649 MB/s
Step 13 Time is 20.923 ms, Bandwidth is 5011.595 MB/s
Step 14 Time is 21.110 ms, Bandwidth is 4967.200 MB/s
Step 15 Time is 21.095 ms, Bandwidth is 4970.732 MB/s
Step 16 Time is 20.885 ms, Bandwidth is 5020.713 MB/s
Step 17 Time is 21.065 ms, Bandwidth is 4977.812 MB/s
Step 18 Time is 21.062 ms, Bandwidth is 4978.521 MB/s
Step 19 Time is 21.094 ms, Bandwidth is 4970.968 MB/s

#################### pagable memory #########################

Data Length is 104857600
========= Test Host To Device ========
Step  0 Time is 8.635 ms, Bandwidth is 12143.324 MB/s
Step  1 Time is 8.582 ms, Bandwidth is 12218.317 MB/s
Step  2 Time is 8.578 ms, Bandwidth is 12224.015 MB/s
Step  3 Time is 8.578 ms, Bandwidth is 12224.015 MB/s
Step  4 Time is 8.578 ms, Bandwidth is 12224.015 MB/s
Step  5 Time is 8.578 ms, Bandwidth is 12224.015 MB/s
Step  6 Time is 8.577 ms, Bandwidth is 12225.440 MB/s
Step  7 Time is 8.577 ms, Bandwidth is 12225.440 MB/s
Step  8 Time is 8.577 ms, Bandwidth is 12225.440 MB/s
Step  9 Time is 8.578 ms, Bandwidth is 12224.015 MB/s
Step 10 Time is 8.578 ms, Bandwidth is 12224.015 MB/s
Step 11 Time is 8.578 ms, Bandwidth is 12224.015 MB/s
Step 12 Time is 8.578 ms, Bandwidth is 12224.015 MB/s
Step 13 Time is 8.578 ms, Bandwidth is 12224.015 MB/s
Step 14 Time is 8.577 ms, Bandwidth is 12225.440 MB/s
Step 15 Time is 8.588 ms, Bandwidth is 12209.781 MB/s
Step 16 Time is 8.584 ms, Bandwidth is 12215.471 MB/s
Step 17 Time is 8.578 ms, Bandwidth is 12224.015 MB/s
Step 18 Time is 8.578 ms, Bandwidth is 12224.015 MB/s
Step 19 Time is 8.578 ms, Bandwidth is 12224.015 MB/s
========= Test Device To Host ========
Step  0 Time is 7.978 ms, Bandwidth is 13143.344 MB/s
Step  1 Time is 7.962 ms, Bandwidth is 13169.756 MB/s
Step  2 Time is 7.961 ms, Bandwidth is 13171.411 MB/s
Step  3 Time is 7.957 ms, Bandwidth is 13178.032 MB/s
Step  4 Time is 7.950 ms, Bandwidth is 13189.635 MB/s
Step  5 Time is 7.957 ms, Bandwidth is 13178.032 MB/s
Step  6 Time is 7.955 ms, Bandwidth is 13181.345 MB/s
Step  7 Time is 7.957 ms, Bandwidth is 13178.032 MB/s
Step  8 Time is 7.949 ms, Bandwidth is 13191.295 MB/s
Step  9 Time is 7.949 ms, Bandwidth is 13191.295 MB/s
Step 10 Time is 7.949 ms, Bandwidth is 13191.295 MB/s
Step 11 Time is 7.949 ms, Bandwidth is 13191.295 MB/s
Step 12 Time is 7.949 ms, Bandwidth is 13191.295 MB/s
Step 13 Time is 7.949 ms, Bandwidth is 13191.295 MB/s
Step 14 Time is 7.949 ms, Bandwidth is 13191.295 MB/s
Step 15 Time is 7.949 ms, Bandwidth is 13191.295 MB/s
Step 16 Time is 7.948 ms, Bandwidth is 13192.954 MB/s
Step 17 Time is 7.949 ms, Bandwidth is 13191.295 MB/s
Step 18 Time is 7.949 ms, Bandwidth is 13191.295 MB/s
Step 19 Time is 7.950 ms, Bandwidth is 13189.635 MB/s

*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>

// step 1. compile and run
// step 2. comment this line, compile and run
#define USE_PINNED_MEMORY

#define TIME(a,b) ((double)((b).tv_sec-(a).tv_sec) * 1000.0 + (double)((b).tv_usec-(a).tv_usec)/(double)1000.0)

#define cudaErrCheck(stat) { cudaErrCheck_((stat), __FILE__, __LINE__); }
void cudaErrCheck_(hipError_t stat, const char *file, int line) {
   if (stat != hipSuccess) {
      fprintf(stderr, "CUDA Error: %s %s %d\n", hipGetErrorString(stat), file, line);
   }
}


__global__ void function()
{
    int tid = threadIdx.x;
    int gid = blockIdx.x;
    printf("Thread Id: %2d, Block Id: %2d Print\n", tid, gid);
}

int main(int argc, char* argv[])
{
    size_t length = 1024*1024*100;

    // use 1st arg in cli as length
    if (argc > 1) length = atoi(argv[1]);
    printf("Data Length is %lld\n", (long long)length);

    char *host = NULL;
    char *device = NULL;

    // malloc device memory
    cudaErrCheck(hipMalloc((void**)&device, length));

    // malloc host memory
#if defined(USE_PINNED_MEMORY)
    cudaErrCheck(hipHostMalloc((void**)&host, length, hipHostMallocDefault));
#else
    host = (char*)malloc(length);
#endif

    // init array in host memory
    for (int i = 0; i < length; i ++) host[i] = 100;

    printf("========= Test Host To Device ========\n");
    for (int s = 0; s < 20; s ++)  // repeat 20 times
    {
        struct timeval t0, t1;
        gettimeofday(&t0, NULL);
        cudaErrCheck(hipMemcpy(device, host, length, hipMemcpyHostToDevice));
        gettimeofday(&t1, NULL);
        printf("Step %2d Time is %.3lf ms, Bandwidth is %.3lf MB/s\n",
            s, TIME(t0, t1), length / TIME(t0, t1) / 1000.0);
    }

    printf("========= Test Device To Host ========\n");
    for (int s = 0; s < 20; s ++)  // repeat 20 times
    {
        struct timeval t0, t1;
        gettimeofday(&t0, NULL);
        cudaErrCheck(hipMemcpy(host, device, length, hipMemcpyDeviceToHost));
        gettimeofday(&t1, NULL);
        printf("Step %2d Time is %.3lf ms, Bandwidth is %.3lf MB/s\n",
            s, TIME(t0, t1), length / TIME(t0, t1) / 1000.0);
    }

    cudaErrCheck(hipFree(device));
#if defined(USE_PINNED_MEMORY)
    cudaErrCheck(hipHostFree(host));
#else
    free(host);
#endif

    return 0;
}
