
#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>

#define N (1024 * 1024 * 1024)  // 1G elements

#define elementsPerThread 8

// Naive reduction kernel
__global__ void reduce_optimized(float* d_in, float* d_out) {
    // Shared memory declaration
    // - Dynamic shared memory allocation using extern keyword
    // - Visible to all threads in the block
    extern __shared__ float sdata[];
    
    // each thread loads one element GMEM -> SMEM
    // sync to Ensures all threads finish loading before computation
    unsigned int tid = threadIdx.x;  // Thread ID within block
    unsigned int i = (blockIdx.x * blockDim.x + tid) * elementsPerThread;  // Global index

    float sum = 0.0f;
    #pragma unroll
    for (size_t ii = 0; ii < elementsPerThread; ii++) {
        int gtid = i + ii;
        if (gtid < N) {
            sum += d_in[gtid];
        }
    }
    sdata[tid] = sum;
    __syncthreads();

    // do reduction in shared mem
    // - Sequential addressing: Stride starts at half block size
    // - Avoids thread divergence within warps (tid < s condition)
    for(unsigned int s = blockDim.x/2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        // Prevents race conditions during shared memory access
        __syncthreads();
    }

    // write result for this block to global mem & acc using atomicAdd
    if (tid == 0) atomicAdd(d_out, sdata[0]);
}

// CPU reference implementation (baseline for validation)
float reduce_cpu(float* data, size_t n) {
    // float sums up to 16777216 (2^24)
    double sum = 0.0f;
    size_t i = 0;
    for(i = 0; i < n; ++i) sum += data[i];

    // printf("n: %ld, i: %ld, sum: %.2f\n", n, i, (float)sum);
    return (float)sum;
}

int main() {
    float *h_in, *h_out;
    float *d_in, *d_out;

    // Host allocations and initialize
    h_in = new float[N];
    h_out = new float[1];
    h_out[0] = 0.0f;

    for(int i = 0; i < N; ++i) h_in[i] = 1.0f;  // (1.0f) for easier validation

    // Device allocations
    hipMalloc(&d_in, N * sizeof(float));
    hipMalloc(&d_out, sizeof(float));
    
    // Host-to-device data transfer
    hipMemcpy(d_in, h_in, N * sizeof(float), hipMemcpyHostToDevice);
    // Initialize output on device
    hipMemset(d_out, 0, sizeof(float));

    // Launch config
    const int block_size = 512;  // Typical starting value
    const int grid_size = (N + block_size - 1) / block_size / elementsPerThread;
    std::cout << "N: " << N/1024/1024 << "M, grid_size: " << grid_size << ", block_size: " << block_size << ", elementsPerThread: " << elementsPerThread << std::endl;
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    hipEventRecord(start);
    // <<<grid, block, shared_mem>>> - Explicit shared memory size specification
    reduce_optimized<<<grid_size, block_size, block_size*sizeof(float)>>>(d_in, d_out);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // Copy result back to host
    hipMemcpy(h_out, d_out, sizeof(float), hipMemcpyDeviceToHost);

    // Result validation
    float cpu_result = reduce_cpu(h_in, N);
    float gpu_result = h_out[0];
    float diff = fabs(cpu_result - gpu_result);
    printf("diff: %2.f, CPU: %.1f, GPU: %.1f.\n", diff, cpu_result, gpu_result);

    float time_ms;
    hipEventElapsedTime(&time_ms, start, stop);
    const size_t data_size = N * sizeof(float);
    // 2.0f * data_size if copy in and out
    float bandwidth = (1.0f * data_size) / (time_ms / 1000.0f) / (1e9);  // GB/s
    const char* success = (diff < 1e-5) ? "Success" : "Error";
    printf("%s! time cost: %.3f ms, HBM Bandwidth: %.2f GB/s\n", success, time_ms, bandwidth);

    // clean up
    hipFree(d_in);
    hipFree(d_out);
    delete[] h_in;
    delete[] h_out;
    return 0;
}
