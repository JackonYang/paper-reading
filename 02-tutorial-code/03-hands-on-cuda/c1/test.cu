/*
CUDA hello world.
Run a device function in multi threads.

`make run`

output:

Thread Id:  0 Print
Thread Id:  1 Print
Thread Id:  2 Print
Thread Id:  3 Print
Thread Id:  4 Print
Thread Id:  5 Print
Thread Id:  6 Print
Thread Id:  7 Print

 */


#include <hip/hip_runtime.h>
#include <stdio.h>


__global__ void kernel_func()
{
    int tid = threadIdx.x;
    printf("Thread Id: %2d Print\n", tid);
}

int main(int argc, char* argv[])
{
    // https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#kernels
    // <<<GirdSize, BlockSize>>>, 1 block, 8 thread per block.
    kernel_func<<<1, 8>>>();
    hipDeviceSynchronize();
    return 0;
}

