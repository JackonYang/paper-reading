/*
CUDA hello world.
Add two vectors in parallel using CUDA.

`make run`
*/


#include <hip/hip_runtime.h>
#include <iostream>

#define N 8192

// CUDA kernel for vector addition
__global__ void vector_add(const float *a, const float *b, float *c)
{
    // global thread index
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < N)  // check the bound
    {
        c[tid] = a[tid] + b[tid];
    }
}

// Host function to add two vectors
int main(int argc, char* argv[])
{
    float *a, *b, *c;  // host copies of a, b, c
    float *dev_a, *dev_b, *dev_c;  // device copies of a, b, c

    // Allocate memory on the host and initialize
    a = (float*)malloc(N * sizeof(float));
    b = (float*)malloc(N * sizeof(float));
    c = (float*)malloc(N * sizeof(float));

    for (int i = 0; i < N; i++)
    {
        a[i] = static_cast<float>(i);
        b[i] = static_cast<float>(i * 2);
    }

    // Allocate memory on the device
    hipMalloc((void**)&dev_a, N * sizeof(float));
    hipMalloc((void**)&dev_b, N * sizeof(float));
    hipMalloc((void**)&dev_c, N * sizeof(float));

    // Copy input data from host to device
    hipMemcpy(dev_a, a, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, N * sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel
    int block_size = 256;
    int grid_size = (N + block_size - 1) / block_size;
    std::cout << "grid_size: " << grid_size << ", block_size: " << block_size << std::endl;

    vector_add<<<grid_size, block_size>>>(dev_a, dev_b, dev_c);

    // Copy the result from device to the host
    hipMemcpy(c, dev_c, N * sizeof(float), hipMemcpyDeviceToHost);

    // clean up (free device memory)
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);


    // Verify the result
    for (int i = 0; i < N; i++)
    {
        if (c[i] != a[i] + b[i])
        {
            std::cerr << "Error: mismatch at " << i << " expected " << a[i] + b[i] << " but got " << c[i] << std::endl;
            return 1;
        }
    }

    // print first 10 elements of the result
    for (int i = 0; i < 10; i++)
    {
        std::cout << c[i] << " ";
    }
    std::cout << std::endl;

    std::cout << "Success!" << std::endl;

    free(a);
    free(b);
    free(c);

    return 0;
}

