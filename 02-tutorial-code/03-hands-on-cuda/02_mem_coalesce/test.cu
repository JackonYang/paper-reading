/*
CUDA hello world.
Add two vectors in parallel using CUDA.

`make run`
*/


#include <hip/hip_runtime.h>
#include <iostream>

#define N 1024 * 1024 * 1024
#define elementsPerThread 4

// CUDA kernel for vector addition
__global__ void vector_add(const float *a, const float *b, float *c)
{
    // hit peak bw at elementsPerThread = 2
    // int base = (blockIdx.x * blockDim.x + threadIdx.x) * elementsPerThread;
    // #pragma unroll
    // for (int i = 0; i < elementsPerThread; ++i) {
    //     int tid = base + i;
    //     if (tid < N) {
    //         c[tid] = a[tid] + b[tid];
    //     }
    // }

    // impl 2: using float4 instruction
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < N/4) {
        float4 a4 = reinterpret_cast<const float4*>(a)[tid];
        float4 b4 = reinterpret_cast<const float4*>(b)[tid];
        float4 c4;
        c4.x = a4.x + b4.x;
        c4.y = a4.y + b4.y;
        c4.z = a4.z + b4.z;
        c4.w = a4.w + b4.w;
        reinterpret_cast<float4*>(c)[tid] = c4;
    }
}

// Host function to add two vectors
int main(int argc, char* argv[])
{
    float *a, *b, *c;  // host copies of a, b, c
    float *dev_a, *dev_b, *dev_c;  // device copies of a, b, c

    // Allocate memory on the host and initialize
    a = (float*)malloc(N * sizeof(float));
    b = (float*)malloc(N * sizeof(float));
    c = (float*)malloc(N * sizeof(float));

    for (int i = 0; i < N; i++)
    {
        a[i] = static_cast<float>(i);
        b[i] = static_cast<float>(i * 2);
    }

    // Allocate memory on the device
    hipMalloc((void**)&dev_a, N * sizeof(float));
    hipMalloc((void**)&dev_b, N * sizeof(float));
    hipMalloc((void**)&dev_c, N * sizeof(float));

    // Copy input data from host to device
    hipMemcpy(dev_a, a, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, N * sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel
    int block_size = 256;
    int grid_size = (N + block_size - 1) / block_size / elementsPerThread;
    std::cout << "N: " << N/1024/1024 << "M, grid_size: " << grid_size << ", block_size: " << block_size << ", elementsPerThread: " << elementsPerThread << std::endl;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    vector_add<<<grid_size, block_size>>>(dev_a, dev_b, dev_c);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float time_ms;
    hipEventElapsedTime(&time_ms, start, stop);

    // Copy the result from device to the host
    hipMemcpy(c, dev_c, N * sizeof(float), hipMemcpyDeviceToHost);

    // clean up (free device memory)
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    // print first 10 elements of the result
    // for (int i = 0; i < 10; i++)
    // {
    //     std::cout << c[i] << " ";
    // }
    // std::cout << std::endl;

    // Verify the result
    for (int i = 0; i < N; i++)
    {
        if (c[i] != a[i] + b[i])
        {
            std::cerr << "Error: mismatch at " << i << " expected " << a[i] + b[i] << " but got " << c[i] << std::endl;
            return 1;
        }
    }

    const size_t data_size = N * sizeof(float);
    float bandwidth = (2.0f * data_size) / (time_ms / 1000.0f) / (1e9);  // GB/s
    printf("Success! time cost: %.3f ms, HBM Bandwidth: %.2f GB/s\n", time_ms, bandwidth);

    free(a);
    free(b);
    free(c);

    return 0;
}

