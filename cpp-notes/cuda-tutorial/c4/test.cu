/*
matrix add using CUDA
demonstrate that GPU cache line is important to high speed performance

compile: make

run: ./test 10240 1024

output(using 2080ti):

#################### Better Cache #########################
<M, N> = <10240 1024>
Take 0.247000 ms 169.809879 GB/s
Take 0.239000 ms 175.493891 GB/s
Take 0.239000 ms 175.493891 GB/s
Take 0.239000 ms 175.493891 GB/s
Take 0.240000 ms 174.762667 GB/s
Take 0.239000 ms 175.493891 GB/s
Take 0.239000 ms 175.493891 GB/s
Take 0.240000 ms 174.762667 GB/s
Take 0.238000 ms 176.231261 GB/s
Take 0.239000 ms 175.493891 GB/s
Take 0.239000 ms 175.493891 GB/s
Take 0.239000 ms 175.493891 GB/s
Take 0.238000 ms 176.231261 GB/s
Take 0.238000 ms 176.231261 GB/s
Take 0.239000 ms 175.493891 GB/s
Take 0.239000 ms 175.493891 GB/s
Take 0.239000 ms 175.493891 GB/s
Take 0.238000 ms 176.231261 GB/s
Take 0.239000 ms 175.493891 GB/s
Take 0.240000 ms 174.762667 GB/s

#################### Worse Cache #########################
<M, N> = <10240 1024>
Take 0.760000 ms 55.188211 GB/s
Take 0.752000 ms 55.775319 GB/s
Take 0.768000 ms 54.613333 GB/s
Take 0.763000 ms 54.971219 GB/s
Take 0.767000 ms 54.684537 GB/s
Take 0.751000 ms 55.849587 GB/s
Take 0.748000 ms 56.073583 GB/s
Take 0.752000 ms 55.775319 GB/s
Take 0.750000 ms 55.924053 GB/s
Take 0.751000 ms 55.849587 GB/s
Take 0.750000 ms 55.924053 GB/s
Take 0.750000 ms 55.924053 GB/s
Take 0.761000 ms 55.115690 GB/s
Take 0.751000 ms 55.849587 GB/s
Take 0.765000 ms 54.827503 GB/s
Take 0.768000 ms 54.613333 GB/s
Take 0.751000 ms 55.849587 GB/s
Take 0.766000 ms 54.755927 GB/s
Take 0.750000 ms 55.924053 GB/s
Take 0.766000 ms 54.755927 GB/s

*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hiprand/hiprand.h>
#include <sys/time.h>
#include <assert.h>
#include <iostream>
#include <limits>
#include <mma.h>
#include <vector>


// step 1. compile and run. fast
// step 2. comment this line, compile and run. slow
#define BETTER_CACHE

#define TIME(a,b) ((double)((b).tv_sec-(a).tv_sec) * 1000.0 + (double)((b).tv_usec-(a).tv_usec)/(double)1000.0)

// Define some error checking macros.
#define cudaErrCheck(stat) { cudaErrCheck_((stat), __FILE__, __LINE__); }
void cudaErrCheck_(hipError_t stat, const char *file, int line) {
   if (stat != hipSuccess) {
      fprintf(stderr, "CUDA Error: %s %s %d\n", hipGetErrorString(stat), file, line);
   }
}


__global__ void AddGpuKernel(float *C,
    const float *A, const float *B,
    size_t M, size_t N)
{
    const int tidx = threadIdx.x + blockIdx.x * blockDim.x;
    const int tidy = threadIdx.y + blockIdx.y * blockDim.y;
    const int tnumx = blockDim.x * gridDim.x;
    const int tnumy = blockDim.y * gridDim.y;

#if defined(BETTER_CACHE)
    for (int i = tidy; i < M; i += tnumy)
    {
        for (int j = tidx; j < N; j += tnumx)
        {
            const int offset = i * N + j;
            C[offset] = A[offset] + B[offset];
        }
    }
#else
    for (int i = tidy; i < N; i += tnumy)
    {
        for (int j = tidx; j < M; j += tnumx)
        {
            const int offset = j * N + i;
            C[offset] = A[offset] + B[offset];
        }
    }
#endif
}


int main(int argc, char* argv[]) {

    int M = atoi(argv[1]);
    int N = atoi(argv[2]);
    printf("<M, N> = <%d %d>\n", M, N);

    // cudaSetDevice(0);

    // input data
    float *A_device;
    float *B_device;

    // output data
    float *C_device;

    // init input data
    size_t size = M * N * sizeof(float);
    cudaErrCheck(hipMalloc((void**)(&A_device), size));
    cudaErrCheck(hipMalloc((void**)(&B_device), size));
    cudaErrCheck(hipMemset(A_device, 0, size));
    cudaErrCheck(hipMemset(B_device, 0, size));

    // init output data
    cudaErrCheck(hipMalloc((void**)(&C_device), size));

    // used to log processing speed
    size_t bytes = M * N * sizeof(float);

    for (int s = 0; s < 20; s ++)
    {
        dim3 grid(32, 32, 1);
        dim3 block(32, 32, 1);

        struct timeval t0, t1;
        gettimeofday(&t0, NULL);

        AddGpuKernel<<<grid, block>>>(C_device, A_device, B_device, M, N);
        hipDeviceSynchronize();

        gettimeofday(&t1, NULL);
        printf("Take %.6f ms %.6lf GB/s\n", TIME(t0, t1), bytes / TIME(t0, t1) / 1000000.0);
    }

    cudaErrCheck(hipFree(A_device));
    cudaErrCheck(hipFree(B_device));
    cudaErrCheck(hipFree(C_device));

    return 0;
}
