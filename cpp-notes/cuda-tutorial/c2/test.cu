/*
passing parameters to device function.
1. demonstrate data copying between host(CPU memory) and device(GPU memory)
2. understand why memory copy in CUDA is very important

compile: make

run: ./test

output:
2 + 7 = 9
 */


#include <hip/hip_runtime.h>
#include <stdio.h>


__global__ void add(int a, int b, int *c)
{
    *c = a + b;
}

int main() {
    int c;
    int *dev_c;

    hipMalloc((void**)&dev_c, sizeof(int));
    add<<<1,1>>>(2, 7, dev_c);

    // copy from device to host
    hipMemcpy(&c,
               dev_c,
               sizeof(int),
               hipMemcpyDeviceToHost);

    printf("2 + 7 = %d\n", c);
    hipFree(dev_c);

    return 0;
}
