/*
CUDA hello world.
Run a device function in multi threads.

compile: make

run: ./test

output:

Thread Id:  0 Print
Thread Id:  1 Print
Thread Id:  2 Print
Thread Id:  3 Print
Thread Id:  4 Print
Thread Id:  5 Print
Thread Id:  6 Print
Thread Id:  7 Print

 */


#include <hip/hip_runtime.h>
#include <stdio.h>


__global__ void function()
{
    int tid = threadIdx.x;
    printf("Thread Id: %2d Print\n", tid);
}

int main(int argc, char* argv[])
{
    // https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#kernels
    // Kernel invocation with 8 threads
    function<<<1, 8>>>();
    hipDeviceSynchronize();
    return 0;
}

